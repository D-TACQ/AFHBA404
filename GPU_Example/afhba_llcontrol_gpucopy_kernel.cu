#include "hip/hip_runtime.h"
#include "afhba-llcontrol-gpucopy.h"

// hello

#define NSEC_PER_CLK  1		// SWAG
#define DEBUG_PERIODIC_STATUS 0

__device__ int stop;

__device__ void nsleep(unsigned nsec) {
	long long start_clock = clock64();
	long long clock_count = nsec/NSEC_PER_CLK;
	//	printf("nsleep: nsec:%u clock_count:%llu\n", nsec, clock_count);
	for (unsigned pollcat = 0; clock64() - start_clock  < clock_count; ) {
		if (++pollcat&0xfffff == 0){
			printf("nsleep pc:%u start:%llu now:%llu end:%llu\n", pollcat, start_clock, clock64() - start_clock,  clock_count);
		}
	}
}

__device__ int wait_sample(int ii, unsigned* tlp, unsigned tl0, short* pai0)
{
	unsigned tl;
	unsigned tl0p1 = tl0+1;
	
	for (unsigned pollcat = 0; (tl = *tlp) == tl0; ){
		if ((++pollcat&0x0fffff) == 0){
			printf("ii:%10d pollcat:%08x nothing to see at %p %08x %04x %04x %04x %04x\n",
					ii, pollcat, tlp, *tlp, pai0[0]&0xffff, pai0[1]&0xffff, pai0[2]&0xffff, pai0[3]&0xffff );
			if (ii > 0){
				printf("QUITTING on data flow stop\n");
				stop = 1;
				return 0;
			}
		}else{
			nsleep(1000);
		}
	}
	if (tl0p1 != tl){
		printf("ERROR: wait_sample() %d missing tl tl0:%u wanted:%u got:%u %s\n", ii, tl0, tl0p1, tl, tl0p1 == tl? "EQ": "NE");
		stop = 1;
	}
	return tl;
}

__global__ void llcontrol_gpu_A_matrix(void * volatile ai_buffer_ptr,
		unsigned * volatile ao_buffer_ptr,
		short * total_data,
		float* AMX,
		int nCycles){
	unsigned * tlatch = &((unsigned*)ai_buffer_ptr)[NCHAN/2+1];
	short * pai0 = (short*)ai_buffer_ptr;
	unsigned * pvi = (unsigned*)ai_buffer_ptr;
	short * pao0 = (short*)ao_buffer_ptr;
	int proc_number = blockIdx.x*blockDim.x + threadIdx.x;
	bool proc0 = (proc_number==0);
	int ao = proc_number;

	printf("%d Starting data loop now! %d cycles NCHAN %d blk:%d dim:%d tid:%d\n", proc_number, nCycles, NCHAN, blockIdx.x, blockDim.x, threadIdx.x);

	unsigned tl0 = *tlatch;
	volatile unsigned tl;

	for (int ii = 0; !stop && ii < nCycles; ii++) {
		if (proc0){
			tl = wait_sample(ii, tlatch, tl0, pai0);
		}
		__syncthreads();
		int ao_result = 0;
		
		for (int ai = 0; ai < AI_CHAN; ++ai){
			ao_result += AMX[ao*AI_CHAN+ai]*pai0[ai];
		}

		if (ao_result > 0x7fff){
			ao_result = 0x7fff;
		}else if (ao_result < -0x7fff){
			ao_result = -0x7fff;
		}
		pao0[ao] = (short)ao_result;
#if DEBUG_PERIODIC_STATUS     
		if (proc0 && ii%40000 == 0){
			printf("Cycle: %10d tl:%10u tl0 %10u\n", ii, tl, tl0);
			for (int iw = 0; iw < 80; ++iw){
				printf("%08x%c", pvi[iw], iw%16==15? '\n': ' ');
			}
		}
#endif
		__syncthreads();
	}



	return;
}

//  if (proc0) printf("Terminating GPU Kernel.\n");
//  return;
//
//}


void llcontrol_gpu_A_matrix_wrapper(void * volatile ai_buffer_ptr,
		unsigned * volatile ao_buffer_ptr,
		short * total_data,
		float* AMX,
		int nCycles){
	//Wrapper to call the CUDA kernel
	llcontrol_gpu_A_matrix<<<1,AO_CHAN>>>(ai_buffer_ptr, ao_buffer_ptr, total_data, AMX, nCycles);
	return;
}
